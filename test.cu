#include<iostream>
#include<cstdio>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(int *a, int *b, int *c){
    c[0] = a[0] + b[0];
    printf("hello\n");
}

int main() {
    int *a;
    hipMallocManaged(&a, sizeof(int));
    int *b;
    hipMallocManaged(&b, sizeof(int));
    int *c;
    hipMallocManaged(&c, sizeof(int));
    a[0] = 1;
    b[0] = 2;
    c[0] = 0;
    cuda_hello<<<1,1>>>(a, b, c);
    hipDeviceSynchronize();
    std::cout << "c:" << c[0] << std::endl; 
    hipFree(a);
    hipFree(b);
    hipFree(c);
    auto error = hipGetLastError();
    std::cout << hipGetErrorString(error) << "\n";
    return EXIT_SUCCESS;
}